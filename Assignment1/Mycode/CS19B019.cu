/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-1
 * Description: Computation of a matrix C = Kronecker_prod(A, B.T)
 *              where A and B are matrices of dimension (m, n) and
 *              the output is of the dimension (m * n, m * n). 
 * Note: All lines marked in --> should be replaced with code. 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
using namespace std;

ofstream outfile; // The handle for printing the output

__global__ void per_row_AB_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    // --> Complete the kernel ....
    long int i = blockIdx.x;
    long int l = threadIdx.x;
    
    for (long int k = 0; k < n; k++)
    {
        for (long int j = 0; j < n; j++)
        {
        
            long int x = k + n * i;
            long int y = l + m * j;
            C[x * m * n + y] = A[i * n + j] * B[l * n + k];
        }
    }
   
}

__global__ void per_column_AB_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    // --> Complete the kernel ....
    long int k = threadIdx.y;
    long int j = threadIdx.x;
    
    if(k>=n || j>=n){
        return ;
    }
    
    for (long int i = 0; i < m; i++)
    {
        for (long int l = 0; l < m; l++)
        {

            long int x = k + n * i;
            long int y = l + m * j;
            C[x * m * n + y] = A[i * n + j] * B[l * n + k];
        }
    }
   
}

__global__ void per_element_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    // --> Complete the kernel ....
    long int t1 = (blockDim.y) * (blockIdx.x) + threadIdx.y;
    long int t2 = (blockDim.x) * (blockIdx.y) + threadIdx.x;
    
    long int t3 = (t1*t2)/(m*n);
    long int t4 = (t1*t2)%(m*n);

    C[t1 * t2] = A[(t3 / n) * n + (t4/m)] * B[(t4%m) * n + (t3 % n)];
}

/**
 * Prints any 1D array in the form of a matrix
 **/
void printMatrix(long int *arr, long int rows, long int cols, char* filename){
    outfile.open(filename);
    for(long int i = 0; i < rows; i++){
        for(long int j = 0; j < cols; j++){
            outfile<<arr[i * cols + j]<<" ";
        }
        outfile<<"\n";
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    long int m,n;	
    cin>>m>>n;	

    // Host_arrays 
    long int *h_a,*h_b,*h_c;

    // Device arrays 
    long int *d_a,*d_b,*d_c;
	
    // Allocating space for the host_arrays 
    h_a = (long int *) malloc(m * n * sizeof(long int));
    h_b = (long int *) malloc(m * n * sizeof(long int));	
    h_c = (long int *) malloc(m * m * n * n * sizeof(long int));	

    // Allocating memory for the device arrays 
    // --> Allocate memory for A on device 
    // --> Allocate memory for B on device 
    // --> Allocate memory for C on device
    hipMalloc(&d_a,m * n * sizeof(long int));
    hipMalloc(&d_b,m * n * sizeof(long int));
    hipMalloc(&d_c,m * m * n * n * sizeof(long int));

    // Read the input matrix A 
    for(long int i = 0; i < m * n; i++) {
        cin>>h_a[i];
    }

    //Read the input matrix B 
    for(long int i = 0; i < m * n; i++) {
        cin>>h_b[i];
    }

    // Transfer the input host arrays to the device 
    // --> Copy A from Host to Device
    // --> Copy B from Host to Device
    hipMemcpy(d_a, h_a, sizeof(long int) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(long int) * m * n, hipMemcpyHostToDevice);

    long int gridDimx, gridDimy;
    
    // Launch the kernels
    /**
     * Kernel 1 - per_row_AB_kernel
     * To be launched with 1D grid, 1D block
     * Each thread should process a complete row of A, B
     **/

    // --> Set the launch configuration
    dim3 grid1(m, 1, 1);
    dim3 block1(m, 1, 1);

    double starttime = rtclock();

    // --> Launch the kernel
    per_row_AB_kernel<<<grid1, block1>>>(d_a, d_b, d_c, m, n);
    hipDeviceSynchronize();                                                           

    double endtime = rtclock(); 
	printtime("GPU Kernel-1 time: ", starttime, endtime);  

    // --> Copy C from Device to Host
    hipMemcpy(h_c, d_c, sizeof(long int) * m * m *n *n, hipMemcpyDeviceToHost);
    printMatrix(h_c, m * n, m * n,"kernel1.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(long int));

    /**
     * Kernel 2 - per_column_AB_kernel
     * To be launched with 1D grid, 2D block
     * Each thread should process a complete column of  A, B
     **/
    
    // --> Set the launch configuration
    dim3 grid2(1, 1, 1);
    dim3 block2(n, n, 1);
    starttime = rtclock(); 

    // --> Launch the kernel
    per_column_AB_kernel<<<grid2,block2>>>(d_a, d_b, d_c, m, n);
    hipDeviceSynchronize(); 

    endtime = rtclock(); 
  	printtime("GPU Kernel-2 time: ", starttime, endtime);  

    // --> Copy C from Device to Host
    hipMemcpy(h_c, d_c, sizeof(long int) * m * m * n * n, hipMemcpyDeviceToHost);
    printMatrix(h_c, m * n, m * n,"kernel2.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(long int));

    /**
     * Kernel 3 - per_element_kernel
     * To be launched with 2D grid, 2D block
     * Each thread should process one element of the output 
     **/
    gridDimx = ceil(float(n * n) / 16);
    gridDimy = ceil(float(m * m) / 64);
    dim3 grid3(gridDimx,gridDimy,1);
    dim3 block3(64,16,1);

    starttime = rtclock();  

    // --> Launch the kernel
    per_element_kernel<<<grid3, block3>>>(d_a, d_b, d_c, m, n);
    hipDeviceSynchronize();

    endtime = rtclock();  
	printtime("GPU Kernel-3 time: ", starttime, endtime);  

    // --> Copy C from Device to Host
    hipMemcpy(h_c, d_c, sizeof(long int) * m * m * n * n, hipMemcpyDeviceToHost);
    printMatrix(h_c, m * n, m * n,"kernel3.txt");

    return 0;
}
